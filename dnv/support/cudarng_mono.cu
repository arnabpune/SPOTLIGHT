#include "cudarng_mono.h"
#include <random>
#include <stdlib.h>
#include <stddef.h>
#include <iostream>

int cuRNG::MAX_RANDS = (1 << 24);
//int cuRNG::WAIT = 20000;
// Start multiple generators for parallelism
namespace cuRNG
{
    inline void resetIndices(int maxn=-1) 
    {
        for(int i=0;i<ngen;i++) index[i]=i;
        if(maxn!=-1) for(int i=0;i<ngen;i++) max_index[i]=maxn;
    }

    void init(int genc,int seed,int max_n,bool preload)
    {
        if(seed==-1)
        {
            std::random_device rd;
            seed = rd();
        }
        d_rands = (float *)malloc(max_n * sizeof(float));
        ngen=genc;
        //Initialize the generator
        hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);

        //Set the seed
        hiprandSetPseudoRandomGeneratorSeed(gen, seed);

        //Allocate GPU memory for the random numbers
        hipMalloc((void **)&h_rands, max_n * sizeof(float));

        //Initialize multiple generators
        for(int i=0;i<ngen;i++)
        {
            index.push_back(i);
            max_index.push_back(0);
        }
        if(preload) generateRandomNumbers(max_n);
        initialized=true;
    }


    float* generateRandomNumbers(long n)
    {
        lock=true;
        //usleep(WAIT);
        resetIndices(n);
        hiprandGenerateUniform(gen, h_rands, n);
        hipMemcpy(d_rands, h_rands, n * sizeof(float), hipMemcpyDeviceToHost);
        lock=false;
        return d_rands;
    }

    void destroy()
    {
        while(lock) {}
        hiprandDestroyGenerator(gen);
        hipFree(h_rands);
        free(d_rands);
        initialized=false;
    }

    float getRN(int devid,int refill)
    {
        //if(index[devid]%1000000==0) std::cout << std::to_string(index[devid]) + " of " + std::to_string(max_index[devid]) << "\n";
        if(!initialized) {std::cout << "cuRNG not initialized!\n"; exit(1);}
        while(lock) {}
        if(index[devid]>=max_index[devid])
        {
            if(refill==-1) refill=MAX_RANDS;
            if(!lock)
            {
                std::cout << "# Calling CUDA RNG with n="+std::to_string(refill)+"\n";
                generateRandomNumbers(refill);
            }
            while(lock) {}
        }
        float ret = d_rands[index[devid]];
        index[devid]+=ngen;
        return ret;
    }
}

/*int main(int argc,char** argv)
{
    long N = 1 << 26;
    std::cout << "N=" << N << "\n";
    cuRNG::init();
    for(int times=0;times<100;times++)
    {
        float *d_randomNumbers = cuRNG::generateRandomNumbers(N);
        std::cout << d_randomNumbers[0] << "\n";
        free(d_randomNumbers);
    }
    cuRNG::destroy();
    std::cout << "Done!\n";
}*/
